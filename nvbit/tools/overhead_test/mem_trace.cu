/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <map>
#include <sstream>
#include <string>
#include <unordered_set>
#include <unordered_map>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

#define CUDA_SAFECALL(call)                                                 \
    {                                                                       \
        call;                                                               \
        hipError_t err = hipGetLastError();                                 \
        if (hipSuccess != err) {                                           \
            fprintf(                                                        \
                stderr,                                                     \
                "Cuda error in function '%s' file '%s' in line %i : %s.\n", \
                #call, __FILE__, __LINE__, hipGetErrorString(err));        \
            fflush(stderr);                                                 \
            _exit(EXIT_FAILURE);                                            \
        }                                                                   \
    }


struct CTXstate {
  /* context id */
  int id;
};

/* lock */
pthread_mutex_t mutex;

/* map to store context state */
std::unordered_map<hipCtx_t, CTXstate*> ctx_state_map;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_callback_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

/* grid launch id, incremented at every launch */
uint64_t grid_launch_id = 0;

void nvbit_at_init() {
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
  GET_VAR_INT(
      instr_begin_interval, "INSTR_BEGIN", 0,
      "Beginning of the instruction interval where to apply instrumentation");
  GET_VAR_INT(
      instr_end_interval, "INSTR_END", UINT32_MAX,
      "End of the instruction interval where to apply instrumentation");
  GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
  std::string pad(100, '-');
  printf("%s\n", pad.c_str());

  /* set mutex as recursive */
  pthread_mutexattr_t attr;
  pthread_mutexattr_init(&attr);
  pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
  pthread_mutex_init(&mutex, &attr);
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  CTXstate* ctx_state = ctx_state_map[ctx];

  /* Get related functions of the kernel (device function that can be
   * called by the kernel) */
  std::vector<hipFunction_t> related_functions =
    nvbit_get_related_functions(ctx, func);

  /* add kernel itself to the related function vector */
  related_functions.push_back(func);

  /* iterate on function */
  for (auto f : related_functions) {
    /* "recording" function was instrumented, if set insertion failed
     * we have already encountered this function */
    if (!already_instrumented.insert(f).second) {
      continue;
    }

    /* get vector of instructions of function "f" */
    const std::vector<Instr*>& instrs = nvbit_get_instrs(ctx, f);

    uint32_t cnt = 0;
    /* iterate on all the static instructions in the function */
    for (auto instr : instrs) {
      if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
          instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
          instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
        cnt++;
        continue;
      }
      if (verbose) {
        instr->printDecoded();
      }

      if (opcode_to_id_map.find(instr->getOpcode()) ==
          opcode_to_id_map.end()) {
        int opcode_id = opcode_to_id_map.size();
        opcode_to_id_map[instr->getOpcode()] = opcode_id;
        id_to_opcode_map[opcode_id] = std::string(instr->getOpcode());
      }

      int opcode_id = opcode_to_id_map[instr->getOpcode()];
      int mref_idx = 0;
      /* iterate on the operands */
      for (int i = 0; i < instr->getNumOperands(); i++) {
        /* get the operand "i" */
        const InstrType::operand_t* op = instr->getOperand(i);

        if (op->type == InstrType::OperandType::MREF) {
          /* insert call to the instrumentation function with its
           * arguments */
          nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
          /* predicate value */
          nvbit_add_call_arg_guard_pred_val(instr);
          /* opcode id */
          nvbit_add_call_arg_const_val32(instr, opcode_id);
          /* memory reference 64 bit address */
          nvbit_add_call_arg_mref_addr64(instr, mref_idx);
          /* add "space" for kernel function pointer that will be set
           * at launch time (64 bit value at offset 0 of the dynamic
           * arguments)*/
          nvbit_add_call_arg_launch_val64(instr, 0);
          /* add pointer to channel_dev*/
          nvbit_add_call_arg_const_val64(
              instr, (uint64_t) 0);
          mref_idx++;
        }
      }
      cnt++;
    }
  }
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
    const char* name, void* params, hipError_t* pStatus) {
  pthread_mutex_lock(&mutex);

  /* we prevent re-entry on this callback when issuing CUDA functions inside
   * this function */
  if (skip_callback_flag) {
    pthread_mutex_unlock(&mutex);
    return;
  }
  skip_callback_flag = true;

  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  CTXstate* ctx_state = ctx_state_map[ctx];

  if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
      cbid == API_CUDA_cuLaunchKernel) {
    cuLaunchKernel_params* p = (cuLaunchKernel_params*)params;

    /* Make sure GPU is idle */
    // hipDeviceSynchronize();
    // assert(hipGetLastError() == hipSuccess);

    if (!is_exit) {
      /* instrument */
      instrument_function_if_needed(ctx, p->f);

      int nregs = 0;
      CUDA_SAFECALL(
          hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

      int shmem_static_nbytes = 0;
      CUDA_SAFECALL(
          hipFuncGetAttribute(&shmem_static_nbytes,
            HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

      /* get function name and pc */
      const char* func_name = nvbit_get_func_name(ctx, p->f);
      uint64_t pc = nvbit_get_func_addr(p->f);

      /* set grid launch id at launch time */
      nvbit_set_at_launch(ctx, p->f, &grid_launch_id, sizeof(uint64_t));
      /* increment grid launch id for next launch */
      grid_launch_id++;

      /* enable instrumented code to run */
      nvbit_enable_instrumented(ctx, p->f, true);

    }
  }
  skip_callback_flag = false;
  pthread_mutex_unlock(&mutex);
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
  pthread_mutex_lock(&mutex);
  CTXstate* ctx_state = new CTXstate;
  assert(ctx_state_map.find(ctx) == ctx_state_map.end());
  ctx_state_map[ctx] = ctx_state;
  pthread_mutex_unlock(&mutex);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
}
